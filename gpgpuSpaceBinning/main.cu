#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


struct Asteroid{
	Asteroid(){}
	Asteroid(float x1, float y1, float z1): x(x1) , y(y1) , value(z1){}
	float x, y, value;

};

std::ostream& operator<<(std::ostream&out, const Asteroid& ast){
	out<<"Asteroid: ("<<ast.x<<", "<<ast.y<<", "<<ast.value<<")";
	return out;
}

struct Point{
	Point(){}
	Point(float x1, float y1):x(x1),y(y1){}
	Point(int x1, int y1):x(x1),y(y1){}
	float x, y;

	bool operator==(const Point & p2){
		
		return (x==p2.x&&y==p2.y)?true:false;
	}
};

std::ostream& operator<<(std::ostream&out, const Point& pt){
	out<<"Point: ("<<pt.x<<", "<<pt.y<<")";
	return out;
}

//Device variable declarations
__device__ Asteroid * d_asteroids;
__device__ float * d_bins;//Bin x is value, y is direction for path calculation

__constant__ long dc_numAsteroids;
__constant__ float dc_stepSize;
__constant__ int dc_gridx;
int gridXtoDevice= 0;





//Global declarations
long numAsteroids;
float stepSize = 10;

float * h_path;
//__device__ float * d_path;

//Host variable declarations
Asteroid * h_asteroids;
float * h_bins; //Bin x is value, y is direction for path calculation
float * h_compareToGPU;

Point h_ship;
Point h_baseStation;
Point h_gridSize;

Point d_ship;
Point d_baseStation;
Point d_gridSize;

//cuda stuff
hipError_t result;


//--------------------INPUT
void readFile(const char* filename){
	using namespace std;

	ifstream inFile(filename, ios::in | ios::binary);
	//string s = "";
	if(inFile.is_open()){
		inFile.seekg (0, ios::beg);
		//read base station position
		float f;
		inFile.read(reinterpret_cast<char*>(&h_baseStation.x), sizeof(float));
		inFile.read(reinterpret_cast<char*>(&h_baseStation.y), sizeof(float));
		inFile.read(reinterpret_cast<char*>(&f), sizeof(float));

		//get number of asteroids
		int pos = inFile.tellg ();		

		inFile.seekg (0, ios::end);
   		numAsteroids = (long)(((int)inFile.tellg())-pos)/12;   		
    	inFile.seekg (pos, ios::beg);

		//initialize array
		h_asteroids = new Asteroid [numAsteroids];

		int count = 0;

		//populate array
		while(inFile.good()){	
			float xVal;
			inFile.read(reinterpret_cast<char*>(&xVal), sizeof(float));
			float yVal;
			inFile.read(reinterpret_cast<char*>(&yVal), sizeof(float));
			float val;
			inFile.read(reinterpret_cast<char*>(&val), sizeof(float));
			h_asteroids[count++]=Asteroid(xVal, yVal, val);
			//cout<<count-1<<": "<<asteroids[count-1]<<endl;
			//cout<<"Float val: "<<f<<" "<<2*f<<endl;
			

		}
	}

	inFile.close();
}

//--------------------OUTPUT TO CONSOLE
void binPrint(float * a){
	using namespace std;
	for(int i = 0; i<h_gridSize.x; ++i){
		for(int j = 0; j<h_gridSize.y; ++j){
			cout<<a[(int)(i*h_gridSize.x+j)]<<", ";
		}
		cout<<endl;
	}
}

void pathPrint(float * a){
	using namespace std;

	vector<Point> onPath;

	for(int i = 0; i<2*2*h_gridSize.x-1; i+=2){
		float x = h_path[i], y=h_path[i+1];
		onPath.push_back(Point(x,y));
	}

	for(int i = 0; i<h_gridSize.x; ++i){
		for(int j = 0; j<h_gridSize.y; ++j){
			vector<Point>::iterator it = find(onPath.begin(), onPath.end(), Point(i,j));
			if(it!=onPath.end())
				cout<<"1, ";
			else
				cout<<"0, ";
		}
		cout<<endl;
	}
}

//--------------------BIN INITIALIZATION
void binInitialization(){
	using namespace std;

	//calculate grid size
	h_gridSize.x=(int)(h_baseStation.x+stepSize/2)/(int) stepSize + 1;
	h_gridSize.y=(int)(h_baseStation.y+stepSize/2)/(int) stepSize + 1;

	//allocate memory
	h_compareToGPU = new float[int(h_gridSize.x*h_gridSize.y)];
	h_bins = new float[int(h_gridSize.x*h_gridSize.y)];
	h_path = new float[int(2*(2*h_gridSize.x-1))];

	//initialize to 0
	for(int i = 0; i< h_gridSize.x*h_gridSize.y;++i){
		h_bins[i]=0;	
		h_compareToGPU[i]=0;
	}

	for(int i = 0; i< 2*(2*h_gridSize.x-1);++i)
		h_path[i]=0;		
}

void binInitializationTest(){
	using namespace std;

	binInitialization();

	//initialize to 0
	for(int i = 0; i< h_gridSize.x*h_gridSize.y;++i)
		h_bins[i]=1;	

	h_bins[(int)(5*h_gridSize.x+5)]=5;	
}

//--------------------CPU FUNCTIONS

void cpuSequentialBinning(){
	using namespace std;
	Point binId, binPos;
	
	for(int i = 0; i<numAsteroids; ++i){	

		binId.x=(int)(h_asteroids[i].x+stepSize/2)/(int) stepSize;
		binId.y=(int)(h_asteroids[i].y+stepSize/2)/(int) stepSize;

		binPos.x=binId.x*stepSize;
		binPos.y=binId.y*stepSize;

		float deltaX = h_asteroids[i].x-binPos.x;
		float deltaY = h_asteroids[i].y-binPos.y;
		
		if((deltaX*deltaX+deltaY*deltaY)<stepSize*stepSize/4){			
			h_bins[(int)(binId.y*h_gridSize.x+binId.x)]+=h_asteroids[i].value;
		}		
	}	
}

void cpuValuePropagation(){
	using namespace std;
	for(int i = 0; i<h_gridSize.x; ++i){
		for(int j = 0; j<h_gridSize.y; ++j){	
			 
			double sumup   = (j-1>=0)?h_bins[(int)((j-1)*h_gridSize.x+i)]:0;				
			double sumleft = (i-1>=0)?h_bins[(int)(j*h_gridSize.x+(i-1))]:0;						

			//h_bins[(int)(j*h_gridSize.x+i)].y = (((sumleft-sumup)>0)?1:-1);
			h_bins[(int)(j*h_gridSize.x+i)]+= max(sumleft, sumup);

			//cout<<"Direction set: "<<h_bins[(int)(j*h_gridSize.x+i)].y<<endl;
			//cout<<(int)(j*h_gridSize.x+i)<<" max is from: "<<(h_bins[(int)(j*h_gridSize.x+i)].y==-1?"above":"left")<<endl;
		

			
		}
	}	
}

void cpuGetPath(){
	

	using namespace std;

	Point currentPoint;
	Point nextIndex(h_gridSize.x-1, h_gridSize.x-1);
	int count = 0;
	do{		
		double sumup   = (nextIndex.y-1>=0)?h_bins[(int)((nextIndex.y-1)*h_gridSize.x+nextIndex.x)]:0;				
		double sumleft = (nextIndex.x-1>=0)?h_bins[(int)(nextIndex.y*h_gridSize.x+(nextIndex.x-1))]:0;

		int nextDir = (max(sumleft,sumup)==sumleft)?1:-1;
		//h_bins[(int)(j*h_gridSize.x+i)].y = (((sumleft-sumup)>0)?1:-1);
		currentPoint.x=nextIndex.x;
		currentPoint.y=nextIndex.y;

		if(nextDir==1){nextIndex.x-=1;}
		if(nextDir==-1){nextIndex.y-=1;}
		//cout<<"current  : "<<currentPoint<<endl;
		
		h_path[count++]=currentPoint.x;
		h_path[count++]=currentPoint.y;

		//cout<<"direction: "<<((nextDir==1)?"left":"up")<<endl; 
		//cout<<"next     : "<<endl<<endl;
		
	}while(currentPoint.x!=0 || currentPoint.y!=0);
}

//--------------------GPU FUNCTIONS

__global__ void gpuAllocateAsteroidToBin(Asteroid* asteroids, float* bins){

	
	bins[0]=dc_stepSize;
	bins[1]=dc_gridx;
	bins[2]=dc_numAsteroids;
	/*
	int i_x = blockIdx.x * blockDim.x + threadIdx.x;
	int i_y = blockIdx.y * blockDim.y + threadIdx.y;
	

	int pitch = gridDim.x * blockDim.x;
	

	int i = i_y * pitch + i_x;
	
	if(i<dc_numAsteroids){
	
		float binIdx=(int)(asteroids[i].x+dc_stepSize/2)/(int) dc_stepSize;
		float binIdy=(int)(asteroids[i].y+dc_stepSize/2)/(int) dc_stepSize;

		float binPosx=binIdx*dc_stepSize;
		float binPosy=binIdy*dc_stepSize;

		float deltaX = asteroids[i].x-binPosx;
		float deltaY = asteroids[i].y-binPosy;
		
		if((deltaX*deltaX+deltaY*deltaY)<dc_stepSize*dc_stepSize/4){			
			bins[(int)(binIdy*dc_gridx+binIdx)]+=asteroids[i].value;
		}	
	}
	*/
	
	
}

__global__ void gpuPropagateMaxValues(){

	
	//int i = threadIdx.x + blockIdx.x * blockDim.x;
	//int j = threadIdx.y + blockIdx.y * blockDim.y;

	//int index = gridDim.x*j+i;

	int i = threadIdx.x;//*blockIdx.x+ ;
	int j = threadIdx.y;

	//if(index<dc_gridx){



	//}
	
	
	//int idUp = (j-1)*dc_gridx+i;
	//int idLeft = j*dc_gridx+i-1;

	//double sumup   = (j-1>=0)?d_bins[idUp]:0;				
	//double sumleft = (i-1>=0)?d_bins[idLeft]:0;						

	
	//d_bins[id]+= ((sumleft - sumup)>0)?sumleft:sumup;

	//d_bins[id]=id;

	/*
	int id = j*dc_gridx+i;
	int idUp = (j-1)*dc_gridx+i;
	int idLeft = j*dc_gridx+i-1;
	
	if(i+j==level){
		double sumup   = (j-1>=0)?d_bins[idUp]:0;				
		double sumleft = (i-1>=0)?d_bins[idLeft]:0;						

		//d_bins[id].y = (((sumleft-sumup)>0)?1:-1);
		d_bins[id]+= ((sumleft - sumup)>0)?sumleft:sumup;


	}
	*/


	


}

__global__ void gpuFindLocalPath(){

}

void checkError(hipError_t errorBool, std::string message){
	using namespace std;
	
	if(errorBool!=hipSuccess){
		cout<<errorBool<<endl;
		cout<<"Error: ";
	}
	else
		cout<<"Passed: ";
		
	cout<<message<<"  -   "<<hipGetErrorString(errorBool)<<endl;
	//cin.get();
}

void gpuInitialization(){
	//result = hipSetDevice(0);
	result = hipFree(NULL);
	
		
	//result=hipMalloc(&dc_stepSize, sizeof(float));
	//result=hipMalloc(dc_numAsteroids, sizeof(long));
	//result=hipMalloc(&dc_gridx, sizeof(float));
	gridXtoDevice = (int)(h_gridSize.x);
	result = hipMemcpyToSymbol(HIP_SYMBOL(dc_gridx), &gridXtoDevice, sizeof(int),0,hipMemcpyHostToDevice);
	checkError(result, "Copying gridSize to symbol");

	result = hipMemcpyToSymbol(HIP_SYMBOL(dc_stepSize), &stepSize, sizeof(float), 0, hipMemcpyHostToDevice);
	checkError(result, "Copying stepsize to symbol");	

	result = hipMemcpyToSymbol(HIP_SYMBOL(dc_numAsteroids), &numAsteroids, sizeof(long),0, hipMemcpyHostToDevice);
	checkError(result, "Copying num asteroid to symbol");
	

	

	
	

	//Allocations for binning
	result=hipMalloc(&d_asteroids, numAsteroids*sizeof(Asteroid));
	checkError(result, "Allocating Asteroid memory");

	result = hipMemcpy(d_asteroids, h_asteroids, numAsteroids*sizeof(Asteroid), hipMemcpyHostToDevice);
	checkError(result, "Copying asteroid Data");


	result=hipMalloc(&d_bins, h_gridSize.x*h_gridSize.x*sizeof(float));
	checkError(result, "Allocating Bin memory");

	result = hipMemcpy(d_bins, h_bins, h_gridSize.x*h_gridSize.x*sizeof(float), hipMemcpyHostToDevice);
	checkError(result, "Copying bin data to device");
	
}

void gpuParallelBinning(){
	
	int numThreads = numAsteroids;
	int threadsPerBlock = 1024;
	int numBlocks = ceil((float)numAsteroids/1024);
	int cudaGridx = ceil(sqrtf(numBlocks));

	std::cout<<"numblocks: "<<numBlocks<<", gridX: "<<cudaGridx<<std::endl;

	dim3 cudaBlockSize(32,32,1);
	//dim3 cudaGridSize(1024, 1024);
	dim3 cudaGridSize(cudaGridx, cudaGridx,1);

	
	//gpuAllocateAsteroidToBin<<<cudaGridSize,cudaBlockSize>>>(d_asteroids, d_bins);
	gpuAllocateAsteroidToBin<<<1,1>>>(d_asteroids, d_bins);
	hipDeviceSynchronize();
	//checkError();

}

void gpuValuePropagation(){
	/*
	int numThreads = h_gridSize.x;
	int threadsPerBlock = 1024;
	int numBlocks = ceil((float)numThreads/threadsPerBlock);
	int cudaGridx = ceil(sqrtf(numBlocks));

	std::cout<<"numblocks: "<<numBlocks<<", gridX: "<<cudaGridx<<std::endl;

	dim3 cudaBlockSize(32,32,1);	
	dim3 cudaGridSize(cudaGridx, cudaGridx,1);
	*/

	dim3 cudaBlockSize(11,1,1);
	//dim3 cudaGridSize(1024, 1024);
	dim3 cudaGridSize(1,1,1);
	
	//for(int i = 0; i< 2*h_gridSize.x-1;i++){
	int i = 11;
	gpuPropagateMaxValues<<<cudaGridSize,cudaBlockSize>>>();
	hipDeviceSynchronize();
	//}
}

void gpuCopyDataBack(){
	using namespace std;
	result = hipMemcpy(h_compareToGPU, d_bins, (int)(h_gridSize.x*h_gridSize.x*sizeof(float)), hipMemcpyDeviceToHost);
	checkError(result,"Copying bins data back from gpu");
	//cout<<h_compareToGPU[0]<<":"<<h_compareToGPU[1]<<":"<<h_compareToGPU[2]<<endl;

	binPrint(h_compareToGPU);
	unsigned long long testNumAst = 0;

	result = hipMemcpyFromSymbol(&testNumAst, HIP_SYMBOL(dc_numAsteroids), sizeof(long), 0,hipMemcpyDeviceToHost);
	checkError(result,"Copying numast back from gpu");

	cout<<"Numast:"<<testNumAst<<endl;

	hipFree(d_asteroids);
	hipFree(d_bins);//Bin x is value, y is direction for path calculation


}



int main(int argc, char** argv){
	printf("num %i\n", 8);
	using namespace std;
	string fname = "s_data_02.scan";
	

	readFile(fname.c_str());
	cout<<"ShipPosition: "<<h_ship<<endl;
	cout<<"BasePosition: "<<h_baseStation<<endl;


	//binInitialization();
	binInitializationTest();

	//CPU implementation
	
	//cpuSequentialBinning();	
	//cpuValuePropagation();
	binPrint(h_bins);
	//cpuGetPath();
	//cout<<"Printing path now: "<<endl;
	//pathPrint(h_path);



	//GPU implementation
	gpuInitialization();
	gpuParallelBinning();
	//gpuValuePropagation();
	gpuCopyDataBack();
	

	cout<<"run complete"<<endl;

	cin.get();
	return 0;

}